#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "" //blockIdx
#include <conio.h>
#include <string>

//#define N 100000
#define P 24576


//__constant__ int S[1];

__device__ double atomicAddd(double* address, double val)
{
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val +
                __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

//Функция сложения
__global__ void add(double* pi)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    long long  N = 1000000;
    long long S = N / 24576 + 1;
    long long start = id * S;
    long long finish = (id + 1) * S;
    double result=0;
    double s=0;
    double a = 0;
    double b = 1;
    double w = (b - a) / N;

    if (finish > N)
        finish = N;

    for (long long i = start;i < finish;i++)
    {
        s = (w * i + a) + w / 2;
        result += 4 / (1 + s * s) * w;
    }

    //pi[0] += result;
    atomicAddd(&pi[0], result);
}

int main(int argc, char* argv[])
{
    double* dev_pi;


    double pi[1];


    // int s[1] = { N / P + 1 };
     //0 - кол-во элементов выполняемое одним потоком
     //1 - шаг (b-a)/n


    clock_t t;
    t = clock();

    //Выделение памяти на устройстве
    hipMalloc((void**)&dev_pi, sizeof(double));


    //  hipMemcpyToSymbol(HIP_SYMBOL(S), s, sizeof(int));

      //Копируем массивы на устройство
      //hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);


    add << <96, 256 >> > (dev_pi);

    //Хост ожидает завершения работы девайса
    //hipDeviceSynchronize();

    //Получаем результат
    hipMemcpy(&pi, dev_pi, sizeof(double), hipMemcpyHostToHost);

    //Очищаем память на устройстве
    hipFree(dev_pi);




    t = clock() - t;
    printf("\ntime %.3f\n", ((double)t) / CLOCKS_PER_SEC);
    printf("\nresult = %.15f", pi[0]);//*(1 / 1000000));


    getch();
    return 0;
}
